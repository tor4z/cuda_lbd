
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <ctime>

using namespace std;

#define N           100000
#define RADIUS      3
#define BLOCK_SIZE  16


__global__ void stencil_1d(int *in, int *out){
    __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int lindex = threadIdx.x + RADIUS;

    temp[lindex] = in[gindex];
    if(threadIdx.x < RADIUS) {
        temp[lindex - RADIUS] = in[gindex - RADIUS];
        temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
    }

    __syncthreads();

    int result = 0;
    for(int offset = -RADIUS; offset <= RADIUS; offset++)
        result += temp[lindex + offset];
    
    out[gindex] = result;
}


void fill_ints(int *x, int n){
    fill_n(x, n, 1);
}


int main(void) {
    int *in, *out;
    int *d_in, *d_out;
    clock_t start, end;
    int size = (N + 2 * RADIUS) * sizeof(int);

    in = (int*)malloc(size);
    fill_ints(in, N + 2 * RADIUS);

    out = (int*)malloc(size);
    fill_ints(out, N + 2 * RADIUS);

    hipMalloc((void**)&d_in, size);
    hipMalloc((void**)&d_out, size);

    hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
    hipMemcpy(d_out, out, size, hipMemcpyHostToDevice);

    start = clock();
    stencil_1d<<<N/BLOCK_SIZE, BLOCK_SIZE>>>(d_in + RADIUS, d_out + RADIUS);
    end = clock();
    hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

    cout << "CUDA Duration: " << end - start << "ms." << endl;

    free(in);
    free(out);
    hipFree(d_in);
    hipFree(d_out);
    return 0;
}
